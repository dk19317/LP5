/*
4.Write a CUDA Program for :
   1. Addition of two large vectors
   
check nvcc is intstalled or not by nvcc --version

compile: nvcc filename.cu -o filename
run: ./filename

*/

#include<iostream>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#define BLOCK_SIZE 16
using namespace std;

void fill_array(int *arr, int size){
    for(int i = 0; i < size; i++){
        cin >> arr[i];  // Taking input from user
    }
}

void add_cpu(int *arr1, int *arr2, int *result, int size){
    for(int i = 0; i < size; i++){
        result[i] = arr1[i] + arr2[i];
    }
}

void print_array(int *arr, int size){
    for(int i = 0; i < size; i++){
        cout << arr[i] << " ";
    }
    cout << endl;
}

__global__ void add(int *arr1, int *arr2, int *arr3, int size){
    int block_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(block_id < size){
        arr3[block_id] = arr1[block_id] + arr2[block_id];
    }
}

int main(){
    int *arr1_cpu, *arr2_cpu, *result_cpu;
    int size;

    cout << "Enter size of vector: ";
    cin >> size;

    arr1_cpu = new int[size];
    arr2_cpu = new int[size];
    result_cpu = new int[size];

    cout << "Enter elements for Array 1: ";
    fill_array(arr1_cpu, size);
    print_array(arr1_cpu, size);

    cout << "Enter elements for Array 2: ";
    fill_array(arr2_cpu, size);
    print_array(arr2_cpu, size);

    int *arr1_gpu, *arr2_gpu, *result_gpu;
    
    hipMallocManaged(&arr1_gpu, size * sizeof(int));
    hipMallocManaged(&arr2_gpu, size * sizeof(int));
    hipMallocManaged(&result_gpu, size * sizeof(int));

    hipMemcpy(arr1_gpu, arr1_cpu, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2_cpu, size * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 dimGrid((size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    add<<<dimGrid, dimBlock>>>(arr1_gpu, arr2_gpu, result_gpu, size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(result_cpu, result_gpu, size * sizeof(int), hipMemcpyDeviceToHost);
    cout << "GPU result:\n";
    print_array(result_cpu, size);
    cout << "Elapsed Time = " << elapsedTime << " milliseconds" << endl;

    hipFree(arr1_gpu);
    hipFree(arr2_gpu);
    hipFree(result_gpu);

    add_cpu(arr1_cpu, arr2_cpu, result_cpu, size);
    cout << "CPU result:\n";
    print_array(result_cpu, size);

    return 0;
}
